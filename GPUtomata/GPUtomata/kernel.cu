#include "hip/hip_runtime.h"

#include "GPUtomata.cuh"
#define MAX_SIZE 1000

int main();

hipError_t iterateWithCuda(std::vector<int>& newState, std::vector<int>& currentState, unsigned int size, std::vector<int> ruleset);
std::vector<int> ruleTranslation(std::vector<std::vector<int>> transitionRules);
int checkRuleset(std::vector<int> transitionRules);
void setGliderTest(int* currentState, int size);
void printState(int* currentState, int size);
void initField(std::vector<int>& currentState, std::vector<int>& newState, int size, bool gliderTest);
void automatonSetup(int& size, int& runTime, std::vector<int>& ruleSet);

__global__ void iterateRule(int* newState, int* currentState, int size, int* ruleset) {

    /*
    Using block number, block size and thread id to derive grid coordinates. 
    The same principle used in array to grid translation in following loops.
    */
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int orow = int(index / size);
    int ocol = index % size;
    int row;
    int col;
    int sum = 0;
    /*
      This can be done with one statement since the neighborhood is defined by known offsets. 
      However the compiler will probably end up with equivalent result anyway, plus this way it's more intelligeble.    
    */
    for (int x = -1; x < 2; x++)
    {
        row = (orow + x)%size;

        if (row < 0) {
            row = size + row;
        }
        for (int y = -1; y < 2; y++)
        {
            col = (ocol + y)%size;

            if (col == -1) {
                col = size + col;
            }

            sum += currentState[col + (row * size)];
        }
    }
    /*
    Extendable to more states. If I wanted to build a library this would be replaced by loop.
    */
    if (currentState[index] == 0) {
        newState[index] = ruleset[sum];
    }
    else {
        newState[index] = ruleset[sum+9];
    }
}

int main()
{

    int size = 190;
    std::vector<int> currentState;
    std::vector<int> newState;
    std::vector<int> transitionRules;  
    /*
    N*M vector
    N stands for the number of possible states of a cell
    M for the number of transition rules A -> B, where B is any state.
    */

    int runTime = 100000;
    hipError_t cudaStatus;
    std::vector<std::vector<int>> allConfs = { };
    std::vector<int> surviveStates = { 2, 3 };
    std::vector<int> birthStates = { 3 };
    bool random = true;
           
    automatonSetup(size, runTime, transitionRules);

    newState.resize(size * size);
    currentState.resize(size * size);


    initField(currentState, newState, size, !random);

    printState(currentState.data(), size);
    printf("\n-\n");

    while (runTime > 0) {
        //Iterate in parallel
        cudaStatus = iterateWithCuda(newState, currentState, (size * size), transitionRules);

        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "iterateWithCuda failed! Status: %i", cudaStatus);
            return 1;
        }        
        //Print result once each 100 iterations.
        if (runTime % 100 == 0) {
            
            printState(newState.data(), size);
            printf("\n-\n");
        }
        //Copy new state to current state, clean up new state
        for (int i = 0; i < size * size; i++) {
            currentState[i] = newState[i];
            newState[i] = 0;
        }       
        runTime--;        
    }
  
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed! Status: %i", cudaStatus);
        return 1;
    }

    return 0;
}

std::vector<int> ruleTranslation(std::vector<std::vector<int>> transitionRules) {
    /*
    Sets up ruleset vector of size N*M. 
    */

    std::vector<int> ruleset(transitionRules.size()*10);
    int ruleShift = 0;
    //rule setup
    for each (std::vector<int> state in transitionRules)
    {
        for each (int rule in state)
        {          
            ruleset[rule + ruleShift] = 1;
        }
        ruleShift += 10;
    }
    
    return ruleset;
}

int checkRuleset(std::vector<int> transitionRules)
{
    if (transitionRules.size() > 9) {
        return 1;
    }

    for each (int state in transitionRules)
    {
        if (state > 9) {
            return 1;
        }
    }

    return 0;
}

void setGliderTest(int* currentState, int size)
{
    /*
    Standard minimal GOL glider set roughly in center of the field. 
    Won't work in all possible rulesets but it is a good way to test behavior.
    */
    int yStart = int(((size * size) / 2) / size);
    int xStart = (int((size * size) / 2) % size) + 2;

    currentState[xStart + yStart * size] = 1;
    currentState[xStart + (yStart + 1) * size] = 1;
    currentState[xStart + (yStart + 2) * size] = 1;
    currentState[(xStart + 2) + (yStart + 1) * size] = 1;
    currentState[(xStart + 1) + (yStart + 2) * size] = 1;
}

void printState(int* currentState, int size)
{
    for (int i = 0; i < size * size; i++)
    {
        if (i % size == 0) {
            printf("\n");
        }
        if (currentState[i] == 1) {
            printf("%d ", currentState[i]);
        }
        else {
            printf(" ");
        }       
    }
}

void initField(std::vector<int> &currentState, std::vector<int> &newState, int size, bool gliderTest)
{

    for (int i = 0; i < size * size; i++)
    {
        if (rand() % 10 > 6 && !gliderTest) {
            currentState[i] = 1;
        }
        else {
            currentState[i] = 0;
        }
        newState[i] = 0;
    }
    if (gliderTest) {
        setGliderTest(currentState.data(), size);
    }

}

void automatonSetup(int& size, int& runTime, std::vector<int>& ruleSet)
{
    std::vector<std::vector<int>> allConfs = { };
    std::vector<int> surviveStates = { 2, 3 };
    std::vector<int> birthStates = { 3 };
    std::string arg;
    std::string substring;

    while (true)
    {
        printf("Enter field size: ");
        std::getline(std::cin, arg);
        size = std::stoi(arg);
        arg.clear();
        if (size > MAX_SIZE) {
            printf("Too large field size entered. \nPlease enter number < %d!", MAX_SIZE);
            size = 0;
        }
        else if (size <= 0) {
            printf("Too small field size entered. \nPlease enter number > %d!", 0);
            size = 0;
        }
        else {
            break;
        }
    }

    while (true)
    {
        printf(" \nEnter run time: ");
        std::getline(std::cin, arg);
        runTime = std::stoi(arg);

        arg.clear();

        if (runTime < 0) {
            printf("Negative run time entered. \nPlease enter runtime >= 0. ");
            runTime = 0;
        }
        else {
            break;
        }
    }

    while (true) {
        printf(" \nEnter list of all states for 0 -> 1 transition, as number of active cells separated by space char: ");
        std::getline(std::cin, arg);
        std::stringstream argStream (arg);

        while (std::getline(argStream, substring, ' '))
        {
            birthStates.push_back(std::stoi(substring));
        }

        arg.clear();
        argStream.clear();

        if (checkRuleset(birthStates)!=0) {
            printf("Invalid list of states provided. Please make sure you enter at most 9 entries with each element <= 9. ");
        }
        else {
            break;
        }
    }

    while (true) {
        printf(" \nEnter list of all states for 1 -> 1 transition, as number of active cells separated by space char: ");
        std::getline(std::cin, arg);
        std::stringstream argStream(arg);

        while (std::getline(argStream, substring, ' '))
        {
            birthStates.push_back(std::stoi(substring));
        }

        arg.clear();
        argStream.clear();

        if (checkRuleset(surviveStates) != 0) {
            printf("Invalid list of states provided. Please make sure you enter at most 9 entries with each element <= 9. ");
        }
        else {
            break;
        }
    }

    allConfs.push_back(birthStates);
    allConfs.push_back(surviveStates);
    ruleSet = ruleTranslation(allConfs);
}

// Helper function for using CUDA to update state of the field.
hipError_t iterateWithCuda(std::vector<int> &newState, std::vector<int> &currentState, unsigned int size, std::vector<int> ruleset)
{
    int* dev_currentState = 0;
    int* dev_newState = 0;
    int* dev_ruleset = 0;
    int* newStateArr = newState.data();
    hipError_t cudaStatus;

    int blockSize = 32;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two for states, one for ruleset)    .
    cudaStatus = hipMalloc((void**)&dev_currentState, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_newState, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_ruleset, ruleset.size() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_currentState, currentState.data(), size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_ruleset, ruleset.data(), ruleset.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
    iterateRule << <numBlocks, blockSize >> > (dev_newState, dev_currentState, int(sqrt(size)), dev_ruleset);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "iterateWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching iterateRule!\n", cudaStatus);
        goto Error;
    }

    // Copy new state of the field from GPU buffer to host memory.
    cudaStatus = hipMemcpy(newStateArr, dev_newState, size * sizeof(int), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_currentState);
    hipFree(dev_newState);
    hipFree(dev_ruleset);
    return cudaStatus;
}
